#include "hip/hip_runtime.h"
#include <cstdint>
#include <iostream>
#include "Xdevice/runtime.hpp"

#if (!defined TARGET_CUDA && !defined TARGET_HIP)
#  error "Must define either TARGET_CUDA or TARGET_HIP."
#endif

#ifndef FIB
# define FIB 64
#endif
#define HOSTNAME_SIZE 80

__device__ __forceinline__ void fibonacci(float * __restrict__ buffer)
{
  /* Computes the first FIB fibonacci numbers */
  for (int idx = 0;  idx < FIB; idx++)
  {
    if (idx < 2)
    {
      buffer[idx] = idx;
    }
    else
    {
      buffer[idx] = buffer[idx-1]+buffer[idx-2];
    }
  }
}


__global__ void timed_fibonacci(float * __restrict__ buffer, uint64_t * timer)
{
  /* Count the number of clocks it takes to run the fibonacci function */

  // timers
  uint64_t start, end;

  // start the cycle count
  start = XClock64();

  // Do some work
  fibonacci(buffer);

  // end cycle count
  end = XClock64();

  // Diff the timers and write the result back
  end -= start;
  (*timer) = uint64_t(end);
}


int time_fib()
{
  float * d_buffer;
  uint64_t * d_timer;
  uint64_t h_timer;

  // Allocate the device buffers
  XMalloc((void**)&d_buffer, sizeof(float)*FIB);
  XMalloc((void**)&d_timer, sizeof(uint64_t));

  // calculate the Fib numbers
  timed_fibonacci<<<1,1>>>(d_buffer, d_timer);
  XDeviceSynchronize();

  // Copy the timing data back to the host
  XMemcpy(&h_timer, d_timer, sizeof(uint64_t), XMemcpyDeviceToHost);

  // Free the buffers
  XFree(d_buffer);
  XFree(d_timer);

  return (int)h_timer;
}



int main(int argc, char ** argv)
{
  // Get host name
  char nid_name[HOSTNAME_SIZE];
  gethostname(nid_name, HOSTNAME_SIZE);

  // Make sure we've got devices aboard
  int num_devices;
  XGetDeviceCount(num_devices);
  if (num_devices == 0)
  {
    std::cout << "No devices found on host " << nid_name << std::endl;
    return 1;
  }
  else
  {
    printf("[%s] Found %d device(s).\n", nid_name, num_devices);
  }

  // Time the Fib list creation for each device
  for (int i = 0; i < num_devices; i++)
  {
    printf("[device %d] Clocks taken to compute the first %d Fibonacci numbers: %d\n", i, FIB, time_fib());
  }

  return 0;
}
